#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

__global__ void GPU_kernel(int maxelemszam)
{
 int i;
 for (i = 0; i < maxelemszam; ++i)
 {
  printf("%i\n", i);
 }
}

int main(void)
{
 GPU_kernel << < 1, 1 >> > (100);
 hipDeviceSynchronize();
 printf("Vegrehajtas befejezve!\n");
 return 0;
}
