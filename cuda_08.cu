#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

__global__ void ertek_beallitas(int *c, int szam)
{
    c[0] = szam;
}

int main()
{
 int *dev0_adat;
 int *dev1_adat;
 int akt_adat=0;

 hipSetDevice(0);
 hipMalloc((void**)&dev0_adat, 1 * sizeof(int));
 hipMemcpy(dev0_adat, &akt_adat, 1 * sizeof(int), hipMemcpyHostToDevice); 
 hipSetDevice(1);
 hipMalloc((void**)&dev1_adat, 1 * sizeof(int)); 
 hipMemcpy(dev0_adat, &akt_adat, 1 * sizeof(int), hipMemcpyHostToDevice);
 printf("Memoria lefoglalva...\n");

 hipSetDevice(0);
 ertek_beallitas << <1, 1 >> > (dev0_adat,1);
 hipDeviceSynchronize();
 hipSetDevice(1);
 ertek_beallitas << <1, 1 >> > (dev1_adat, 9);
 hipDeviceSynchronize();
 printf("Kernelek ok...\n");

 hipSetDevice(0);
 hipMemcpy(&akt_adat, dev0_adat, 1 * sizeof(int), hipMemcpyDeviceToHost);
 printf("DEV0: %i\n", akt_adat);
 hipSetDevice(1);
 hipMemcpy(&akt_adat, dev1_adat, 1 * sizeof(int), hipMemcpyDeviceToHost);
 printf("DEV1: %i\n", akt_adat);

 hipMemcpyPeer(dev0_adat,0, dev1_adat,1,1*sizeof(int));
 printf("Csere ok...\n");

 hipSetDevice(0);
 hipMemcpy(&akt_adat, dev0_adat, 1 * sizeof(int), hipMemcpyDeviceToHost);
 printf("DEV0: %i\n",akt_adat);
 hipSetDevice(1);
 hipMemcpy(&akt_adat, dev1_adat, 1 * sizeof(int), hipMemcpyDeviceToHost);
 printf("DEV1: %i\n", akt_adat);
 
 hipFree(dev0_adat);
 hipFree(dev1_adat);
 return 0;
}
