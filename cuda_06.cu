#include "hip/hip_runtime.h"

#include <stdio.h>

__global__ void GPU_kernel(int maxelemszam)
{
 int i;
 int startindex = threadIdx.x + (blockIdx.x * blockDim.x);
 int leptek = blockDim.x * gridDim.x;
 for (i = startindex; i < maxelemszam; i += leptek)
 {
  printf("%i\n", i);
 }
}

int main(void)
{
 int szalak_szama = 128;
 int blokkok_szama = (100000 + szalak_szama - 1) / szalak_szama;

 GPU_kernel <<< blokkok_szama, szalak_szama >>> (100000);
 hipDeviceSynchronize();
 printf("Vegrehajtas befejezve!\n");
 return 0;
}
