#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <math.h>
#include <stdio.h>
#include <windows.h>
#include <time.h>
#include <d2d1.h>
#include <d2d1helper.h>
#pragma comment(lib, "d2d1")

//*****double buffering*****
#define KEPERNYO_WIDTH 600
#define KEPERNYO_HEIGHT 400

D2D1_RECT_U display_area;
ID2D1Bitmap *memkeptarolo = NULL;
unsigned int kepadat[KEPERNYO_WIDTH * KEPERNYO_HEIGHT];
//**************************************

//**********STATISZTIKA*******************
int kezdet, vege;
FILE *statfajl;

void writestat(char *szoveg, int ertek);
void meres_start(void);
void meres_end(void);
int getrandom(int maxnum);
//***************************************

//**************PEGAZUS************
#define MAX_OBJ_NUM 1000000
float raw_verticesX[MAX_OBJ_NUM], raw_verticesY[MAX_OBJ_NUM];
int raw_colors[MAX_OBJ_NUM];
int raw_vertices_length;
//*******CUDA*************
float *dev_raw_verticesX, *dev_raw_verticesY;
unsigned int *dev_raw_colors;
unsigned int *dev_kepadat;
//************************
void data_transfer_to_GPU(void);
void D2D_rajzolas(ID2D1HwndRenderTarget* pRT);
__global__ void render_objects(int maxitemcount, float *arrayX, float *arrayY, unsigned int *colorpuffer, unsigned int *puffer);
//************************************

//***********STANDARD WIN32API WINDOWING************
ID2D1Factory* pD2DFactory = NULL;
ID2D1HwndRenderTarget* pRT = NULL;
#define HIBA_00 TEXT("Error:Program initialisation process.")
HINSTANCE hInstGlob;
int SajatiCmdShow;
char szClassName[] = "WindowsApp";
HWND Form1; //Ablak kezeloje
LRESULT CALLBACK WndProc0(HWND, UINT, WPARAM, LPARAM);
//******************************************************

//*****double buffering*****
void create_main_buffer(void);
void CUDA_cleanup_main_buffer(void);
void swap_main_buffer(void);
//**************************************

//*****drawig algorithms*****
__device__ void CUDA_SetPixel(int x1, int y1, int color, unsigned int *puffer);
__device__ void CUDA_DrawLine(int x1, int y1, int x2, int y2, int color, unsigned int *puffer);
__device__ void CUDA_FillTriangle(int x1, int y1, int x2, int y2, int x3, int y3, int color, unsigned int *puffer);
//**************************************

//*********************************
//The main entry point of our program
//*********************************
int WINAPI WinMain(HINSTANCE hInstance, HINSTANCE hPrevInstance, PSTR szCmdLine, int iCmdShow)
{
 static TCHAR szAppName[] = TEXT("StdWinClassName");
 HWND hwnd;
 MSG msg;
 WNDCLASS wndclass0;
 SajatiCmdShow = iCmdShow;
 hInstGlob = hInstance;

 //*********************************
 //Preparing Windows class
 //*********************************
 wndclass0.style = CS_HREDRAW | CS_VREDRAW;
 wndclass0.lpfnWndProc = WndProc0;
 wndclass0.cbClsExtra = 0;
 wndclass0.cbWndExtra = 0;
 wndclass0.hInstance = hInstance;
 wndclass0.hIcon = LoadIcon(NULL, IDI_APPLICATION);
 wndclass0.hCursor = LoadCursor(NULL, IDC_ARROW);
 wndclass0.hbrBackground = (HBRUSH)GetStockObject(LTGRAY_BRUSH);
 wndclass0.lpszMenuName = NULL;
 wndclass0.lpszClassName = TEXT("WIN0");

 //*********************************
 //Registering our windows class
 //*********************************
 if (!RegisterClass(&wndclass0))
 {
  MessageBox(NULL, HIBA_00, TEXT("Program Start"), MB_ICONERROR);
  return 0;
 }

 //*********************************
 //Creating the window
 //*********************************
 Form1 = CreateWindow(TEXT("WIN0"),
  TEXT("CUDA - DIRECT2D"),
  (WS_OVERLAPPED | WS_SYSMENU | WS_THICKFRAME | WS_MAXIMIZEBOX | WS_MINIMIZEBOX),
  50,
  50,
  KEPERNYO_WIDTH,
  KEPERNYO_HEIGHT,
  NULL,
  NULL,
  hInstance,
  NULL);

 //*********************************
 //Displaying the window
 //*********************************
 ShowWindow(Form1, SajatiCmdShow);
 UpdateWindow(Form1);

 //*********************************
 //Activating the message processing for our window
 //*********************************
 while (GetMessage(&msg, NULL, 0, 0))
 {
  TranslateMessage(&msg);
  DispatchMessage(&msg);
 }
 return msg.wParam;
}

//*********************************
//The window's callback funtcion: handling events
//*********************************
LRESULT CALLBACK WndProc0(HWND hwnd, UINT message, WPARAM wParam, LPARAM lParam)
{
 HDC hdc;
 PAINTSTRUCT ps;

 switch (message)
 {
  //*********************************
  //When creating the window
  //*********************************
 case WM_CREATE:
  srand((unsigned)time(NULL));
  D2D1CreateFactory(D2D1_FACTORY_TYPE_SINGLE_THREADED, &pD2DFactory);
  pD2DFactory->CreateHwndRenderTarget(
   D2D1::RenderTargetProperties(),
   D2D1::HwndRenderTargetProperties(
    hwnd, D2D1::SizeU(KEPERNYO_WIDTH, KEPERNYO_HEIGHT)),
   &pRT);
  hipDeviceReset();
  create_main_buffer();
  hipMalloc((void**)&dev_raw_verticesX, MAX_OBJ_NUM * sizeof(float));
  hipMalloc((void**)&dev_raw_verticesY, MAX_OBJ_NUM * sizeof(float));
  hipMalloc((void**)&dev_raw_colors, MAX_OBJ_NUM * sizeof(unsigned int));
  hipMalloc((void**)&dev_kepadat, KEPERNYO_WIDTH * KEPERNYO_HEIGHT * sizeof(unsigned int));

  int i;
  for (i = raw_vertices_length = 0; i < MAX_OBJ_NUM; ++i)
  {
   raw_verticesX[i] = getrandom(KEPERNYO_WIDTH);
   raw_verticesY[i] = getrandom(KEPERNYO_HEIGHT);
   raw_colors[i] = RGB(getrandom(255), getrandom(255), getrandom(255));
   ++raw_vertices_length;
  }
  data_transfer_to_GPU();
  return 0;
  //*********************************
  //to eliminate color flickering
  //*********************************
 case WM_ERASEBKGND:
  return (LRESULT)1;
  //*********************************
  //Repainting the client area of the window
  //*********************************
 case WM_PAINT:
  hdc = BeginPaint(hwnd, &ps);
  EndPaint(hwnd, &ps);
  D2D_rajzolas(pRT);
  return 0;
  //*********************************
  //Closing the window, freeing resources
  //*********************************
 case WM_CLOSE:
  pRT->Release();
  pD2DFactory->Release();
  hipFree(dev_raw_verticesX);
  hipFree(dev_raw_verticesY);
  hipFree(dev_raw_colors);
  hipFree(dev_kepadat);
  DestroyWindow(hwnd);
  return 0;
  //*********************************
  //Destroying the window
  //*********************************
 case WM_DESTROY:
  PostQuitMessage(0);
  return 0;
 }
 return DefWindowProc(hwnd, message, wParam, lParam);
}

//********************************
//PEGAZUS
//********************************
void create_main_buffer(void)
{
 pRT->CreateBitmap(D2D1::SizeU(KEPERNYO_WIDTH, KEPERNYO_HEIGHT),
  D2D1::BitmapProperties(D2D1::PixelFormat(DXGI_FORMAT_B8G8R8A8_UNORM,
   D2D1_ALPHA_MODE_IGNORE)), &memkeptarolo);
}

void CUDA_cleanup_main_buffer(void)
{
 hipMemset(dev_kepadat, 200, KEPERNYO_HEIGHT*KEPERNYO_WIDTH * sizeof(unsigned int));
}

void swap_main_buffer(void)
{
 display_area.left = 0;
 display_area.top = 0;
 display_area.right = KEPERNYO_WIDTH;
 display_area.bottom = KEPERNYO_HEIGHT;
 memkeptarolo->CopyFromMemory(&display_area, kepadat, KEPERNYO_WIDTH * sizeof(unsigned int));
 pRT->BeginDraw();
 pRT->DrawBitmap(memkeptarolo, D2D1::RectF(0.0f, 0.0f, KEPERNYO_WIDTH, KEPERNYO_HEIGHT), 1.0f, D2D1_BITMAP_INTERPOLATION_MODE_NEAREST_NEIGHBOR, NULL);
 pRT->EndDraw();
}

__device__ void CUDA_SetPixel(int x1, int y1, int color, unsigned int *puffer)
{
 puffer[(y1 * KEPERNYO_WIDTH) + x1] = color;
}

__device__ void CUDA_DrawLine(int x1, int y1, int x2, int y2, int color, unsigned int *puffer)
{
 bool flip = false;
 int swap, offset;

 if (abs(x2 - x1) < 2 && abs(y2 - y1) < 2)
 {
  puffer[(y2*KEPERNYO_WIDTH) + x2] = color; return;
 }
 if (abs(x1 - x2) < abs(y1 - y2))
 {
  swap = x1;
  x1 = y1;
  y1 = swap;

  swap = x2;
  x2 = y2;
  y2 = swap;
  flip = true;
 }
 if (x1 > x2)
 {
  swap = x1;
  x1 = x2;
  x2 = swap;

  swap = y1;
  y1 = y2;
  y2 = swap;
 }
 int dx = x2 - x1;
 int dy = y2 - y1;

 int marker1 = abs(dy) * 2;
 int marker2 = 0;
 int y = y1, x;

 if (flip)
 {
  for (x = x1; x <= x2; ++x)
  {
   offset = (x * KEPERNYO_WIDTH);
   puffer[offset + y] = color;
   marker2 += marker1;
   if (marker2 > dx)
   {
    y += (y2 > y1 ? 1 : -1);
    marker2 -= dx * 2;
   }
  }
 }
 else
 {
  for (x = x1; x <= x2; ++x)
  {
   offset = (y * KEPERNYO_WIDTH);
   puffer[offset + x] = color;
   marker2 += marker1;
   if (marker2 > dx)
   {
    y += (y2 > y1 ? 1 : -1);
    marker2 -= dx * 2;
   }
  }
 }
}

__device__ void CUDA_FillTriangle(int x1, int y1, int x2, int y2, int x3, int y3, int color, unsigned int *puffer)
{
 int Ax, Ay, Bx, By, i, j;
 int swapx, swapy, offset, maxoffset = KEPERNYO_HEIGHT * KEPERNYO_WIDTH;
 if (y1 == y2 && y1 == y3) return;

 if (y1 > y2)
 {
  swapx = x1;
  swapy = y1;
  x1 = x2;
  y1 = y2;
  x2 = swapx;
  y2 = swapy;
 }
 if (y1 > y3)
 {
  swapx = x1;
  swapy = y1;
  x1 = x3;
  y1 = y3;
  x3 = swapx;
  y3 = swapy;
 }
 if (y2 > y3)
 {
  swapx = x3;
  swapy = y3;
  x3 = x2;
  y3 = y2;
  x2 = swapx;
  y2 = swapy;
 }
 int t_height = y3 - y1;
 for (i = 0; i < t_height; ++i)
 {
  bool lower_part = i > y2 - y1 || y2 == y1;
  int part_height = lower_part ? y3 - y2 : y2 - y1;
  float alpha = (float)i / t_height;
  float beta = (float)(i - (lower_part ? y2 - y1 : 0)) / part_height;
  Ax = x1 + (x3 - x1)*alpha;
  Ay = y1 + (y3 - y1)*alpha;
  Bx = lower_part ? x2 + (x3 - x2)*beta : x1 + (x2 - x1)*beta;
  By = lower_part ? y2 + (y3 - y2)*beta : y1 + (y2 - y1)*beta;
  if (Ax > Bx)
  {
   swapx = Ax;
   swapy = Ay;
   Ax = Bx;
   Ay = By;
   Bx = swapx;
   By = swapy;
  }

  offset = (y1 + i)*KEPERNYO_WIDTH;
  for (j = Ax; j < Bx; ++j)
  {
   if (offset + j > maxoffset) continue;
   puffer[offset + j] = color;
  }
 }
}

void data_transfer_to_GPU(void)
{
 hipMemcpy(dev_raw_verticesX, raw_verticesX, raw_vertices_length * sizeof(float), hipMemcpyHostToDevice);
 hipMemcpy(dev_raw_verticesY, raw_verticesY, raw_vertices_length * sizeof(float), hipMemcpyHostToDevice);
 hipMemcpy(dev_raw_colors, raw_colors, raw_vertices_length * sizeof(unsigned int), hipMemcpyHostToDevice);
}

void D2D_rajzolas(ID2D1HwndRenderTarget* pRT)
{
 char hibauzenet[256];

 meres_start();
 CUDA_cleanup_main_buffer();
 hipDeviceSynchronize();//opcionálisan elhagyható
 meres_end();
 writestat("Képpuffer törlése: ", vege);
 strcpy_s(hibauzenet, hipGetErrorString(hipGetLastError()));
 SetWindowTextA(Form1, hibauzenet);

 meres_start();
 int szalak_szama = 128;
 int blokkok_szama = (100000 + szalak_szama - 1) / szalak_szama;
 render_objects << <80,8 >> > (raw_vertices_length, dev_raw_verticesX, dev_raw_verticesY, dev_raw_colors, dev_kepadat);
 hipDeviceSynchronize();
 meres_end();
 writestat("Rendereléshez szükséges idő: ", vege);
 strcpy_s(hibauzenet, hipGetErrorString(hipGetLastError()));
 SetWindowTextA(Form1, hibauzenet);

 meres_start();
 hipMemcpy(kepadat, dev_kepadat, KEPERNYO_WIDTH * KEPERNYO_HEIGHT * sizeof(unsigned int), hipMemcpyDeviceToHost);
 strcpy_s(hibauzenet, hipGetErrorString(hipGetLastError()));
 SetWindowTextA(Form1, hibauzenet);
 swap_main_buffer();
 meres_end();
 writestat("Képkocka másolása és megjelenítése: ", vege);
}

__global__ void render_objects(int maxitemcount, float *arrayX, float *arrayY, unsigned int *colorpuffer, unsigned int *puffer)
{
 int i, px, py, tesztcolor;
 int index = (blockIdx.x * blockDim.x) + (threadIdx.x * 2);
 int stride = blockDim.x * gridDim.x;

 for (i = index; i < maxitemcount - 1; i += stride)
 {
  //CUDA_SetPixel(arrayX[i], arrayY[i], colorpuffer[i], puffer);
  CUDA_DrawLine(arrayX[i], arrayY[i], arrayX[i + 1], arrayY[i + 1], colorpuffer[i], puffer);
  //CUDA_FillTriangle(arrayX[i], arrayY[i], arrayX[i + 1], arrayY[i + 1], arrayX[i + 2], arrayY[i + 2], colorpuffer[i], puffer);
 }
}

void meres_start(void)
{
 kezdet = GetTickCount();
}

void meres_end(void)
{
 vege = GetTickCount() - kezdet;
}

void writestat(char *szoveg, int ertek)
{
 statfajl = fopen("statisztika.txt", "at");
 if (statfajl == NULL) return;
 fprintf(statfajl, "%s: ", szoveg);
 fprintf(statfajl, "%i\n", ertek);
 fclose(statfajl);
}

int getrandom(int maxnum)
{
 return (double)rand() / (RAND_MAX + 1) * maxnum;
}
