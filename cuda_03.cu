#include "hip/hip_runtime.h"

#include <stdio.h>

void CPU_kernel(int maxelemszam)
{
 int i;
 for (i = 0; i < maxelemszam; ++i)
 {
  printf("%i\n", i);
 }
}

int main(void)
{
 CPU_kernel(100);
 printf("Vegrehajtas befejezve!\n");
 return 0;
}
