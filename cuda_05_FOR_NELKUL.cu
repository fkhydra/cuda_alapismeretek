#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

__global__ void GPU_kernel(int maxelemszam)
{
 int aktualis_index = threadIdx.x + (blockIdx.x * blockDim.x);
 if (aktualis_index < maxelemszam) printf("%i\n", aktualis_index);
}

int main(void)
{
 GPU_kernel << < 1, 10 >> > (100);
 hipDeviceSynchronize();
 printf("Vegrehajtas befejezve!\n");
 return 0;
}
