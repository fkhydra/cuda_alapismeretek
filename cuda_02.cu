#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime_api.h"
#include <stdio.h>

int main()
{
 int darabszam;
 int dev, driverVersion = 0, runtimeVersion = 0;

 hipGetDeviceCount(&darabszam);
 if (darabszam == 0) printf("Nem talaltam tamogatott CUDA eszkozt!\n");
 else printf("%i darab CUDA eszkozt talaltam...", darabszam);

 for (dev = 0; dev < darabszam; ++dev)
 {
  hipSetDevice(dev);
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, dev);
  printf("\n%d. eszkoz neve: \"%s\"\n", dev, deviceProp.name);

  hipDriverGetVersion(&driverVersion);
  hipRuntimeGetVersion(&runtimeVersion);

  printf("  CUDA meghajto verzioja / Futtato kornyezet verzioja          %d.%d / %d.%d\n",
   driverVersion / 1000, (driverVersion % 100) / 10,
   runtimeVersion / 1000, (runtimeVersion % 100) / 10);

  printf("  CUDA Capability verzio:   %d.%d\n",
   deviceProp.major, deviceProp.minor);

  char msg[256];
  sprintf_s(msg, sizeof(msg),
   "  Osszmemoria:     %.0f MBytes "
   "(%llu bytes)\n",
   static_cast<float>(deviceProp.totalGlobalMem / 1048576.0f),
   (unsigned long long)deviceProp.totalGlobalMem);
  printf("%s", msg);

  printf("  (%2d) Multiprocesszor, (%3d) CUDA mag/MP: %d CUDA mag\n",
   deviceProp.multiProcessorCount,
   _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor),
   _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor) *
   deviceProp.multiProcessorCount);

  printf(
   "  GPU max. orajel:                            %.0f MHz (%0.2f "
   "GHz)\n",
   deviceProp.clockRate * 1e-3f, deviceProp.clockRate * 1e-6f);

  printf("  Memoria orajel:                             %.0f Mhz\n",
   deviceProp.memoryClockRate * 1e-3f);
  printf("  Memoria atviteli sebesseg:                  %d-bit\n",
   deviceProp.memoryBusWidth);
  printf("  Warp merete:                                %d\n",
   deviceProp.warpSize);
  printf("  Szalak max. szama / multiprocessor: %d\n",
   deviceProp.maxThreadsPerMultiProcessor);
  printf("  Szalak max. szama / blokk:  %d\n",
   deviceProp.maxThreadsPerBlock);
  printf("  Blokk maximalis dimenzioi (x,y,z):  (%d, %d, %d)\n",
   deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1],
   deviceProp.maxThreadsDim[2]);
  printf("  Grid maximalis dimenzioi (x,y,z): (%d, %d, %d)\n",
   deviceProp.maxGridSize[0], deviceProp.maxGridSize[1],
   deviceProp.maxGridSize[2]);
 }
 return 0;
}
