#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

__global__ void GPU_kernel(int maxelemszam)
{
 int i;
 int startindex = threadIdx.x;
 int leptek = blockDim.x;
 for (i = startindex; i < maxelemszam; i += leptek)
 {
  printf("%i\n", i);
 }
}

int main(void)
{
 GPU_kernel << < 1, 10 >> > (100);
 hipDeviceSynchronize();
 printf("Vegrehajtas befejezve!\n");
 return 0;
}
